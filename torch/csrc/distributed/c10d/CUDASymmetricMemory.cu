#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory.hpp>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/cuda/utils.hpp>

#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/error.h>

#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
#include <c10/cuda/driver_api.h>
#endif

#include <sys/socket.h>
#include <sys/syscall.h>
#include <sys/un.h>
#include <unistd.h>

#if defined(CUDART_VERSION) && CUDART_VERSION >= 12030
#define CUDART_SUPPORTS_MULTICAST
#endif

namespace {

bool device_has_multicast_support(int device_idx) {
  if (c10::utils::check_env("TORCH_SYMM_MEM_DISABLE_MULTICAST") == true) {
    return false;
  }
  return c10d::cuda::deviceSupportsMulticast(device_idx);
}

bool allow_overlapping_devices() {
  return c10::utils::check_env("TORCH_SYMM_MEM_ALLOW_OVERLAPPING_DEVICES") ==
      true;
}

class IpcChannel {
 public:
  IpcChannel() : socket_name_(get_socket_name(getpid())) {
    TORCH_CHECK(
        (socket_ = socket(AF_UNIX, SOCK_DGRAM, 0)) != 0,
        "Failed to create socket: ",
        c10::utils::str_error(errno));

    struct sockaddr_un addr = {.sun_family = AF_UNIX};
    std::copy(socket_name_.begin(), socket_name_.end(), addr.sun_path);

    TORCH_CHECK(
        bind(socket_, (struct sockaddr*)&addr, SUN_LEN(&addr)) == 0,
        "Failed to bind socket: ",
        c10::utils::str_error(errno));
  }

  ~IpcChannel() {
    close(socket_);
    unlink(socket_name_.c_str());
  }

  void send_fd(int dst_pid, int fd) {
    struct sockaddr_un addr = {.sun_family = AF_UNIX};
    auto socket_name = get_socket_name(dst_pid);
    std::copy(socket_name.begin(), socket_name.end(), addr.sun_path);

    struct iovec io = {.iov_base = (void*)("fd"), .iov_len = 2};

    char cbuf[CMSG_SPACE(sizeof(int))];
    memset(cbuf, 0, sizeof(cbuf));

    struct msghdr msg {
      .msg_name = (void*)&addr, .msg_namelen = sizeof(struct sockaddr_un),
      .msg_iov = &io, .msg_iovlen = 1, .msg_control = cbuf,
      .msg_controllen = sizeof(cbuf)
    };

    auto cmsg = CMSG_FIRSTHDR(&msg);
    cmsg->cmsg_len = CMSG_LEN(sizeof(int));
    cmsg->cmsg_level = SOL_SOCKET;
    cmsg->cmsg_type = SCM_RIGHTS;

    if (fd != -1) {
      std::copy(
          reinterpret_cast<const char*>(&fd),
          reinterpret_cast<const char*>(&fd) + sizeof(fd),
          reinterpret_cast<char*>(CMSG_DATA(cmsg)));
    } else {
      msg.msg_controllen = 0;
    }

    TORCH_CHECK(
        sendmsg(socket_, &msg, 0) > 0, "Failed to send fd: ", c10::utils::str_error(errno));
  }

  int recv_fd() {
    char buf[2];
    struct iovec io = {.iov_base = (void*)buf, .iov_len = sizeof(buf)};

    char cbuf[CMSG_SPACE(sizeof(int))];
    memset(cbuf, 0, sizeof(cbuf));

    struct msghdr msg = {
        .msg_iov = &io,
        .msg_iovlen = 1,
        .msg_control = cbuf,
        .msg_controllen = sizeof(cbuf)};

    TORCH_CHECK(
        recvmsg(socket_, &msg, 0) > 0,
        "Failed to receive fd: ",
        c10::utils::str_error(errno));

    if (msg.msg_controllen == 0) {
      return -1;
    }

    auto cmsg = CMSG_FIRSTHDR(&msg);
    TORCH_CHECK(cmsg != NULL);
    TORCH_CHECK(cmsg->cmsg_len == CMSG_LEN(sizeof(int)));
    TORCH_CHECK(
        cmsg->cmsg_level == SOL_SOCKET && cmsg->cmsg_type == SCM_RIGHTS);
    return *reinterpret_cast<int*>(CMSG_DATA(cmsg));
  }

  std::vector<int> all_gather_fds(
      int rank,
      const std::vector<int>& pids,
      int fd) {
    size_t world_size = pids.size();
    std::vector<int> fds(pids.size());
    fds[rank] = fd;

    int dst_rank = (rank + 1) % world_size;
    for (size_t step = 1; step < world_size; ++step) {
      int src_rank = (rank + world_size - step) % world_size;
      send_fd(pids[dst_rank], fd);
      fd = recv_fd();
      fds[src_rank] = fd;
    }
    return fds;
  }

  int broadcast_fds(
      int rank,
      int src_rank,
      const std::vector<int>& pids,
      int fd) {
    size_t world_size = pids.size();

    if (rank == src_rank) {
      for (int dst_rank = 0; dst_rank < (int)world_size; ++dst_rank) {
        if (dst_rank == rank) {
          continue;
        }
        send_fd(pids[dst_rank], fd);
      }
      return fd;
    }
    return recv_fd();
  }

 private:
  static std::string get_socket_name(int pid) {
    const char* tmp_dir = "/tmp";
    for (const char* env_var : {"TMPDIR", "TMP", "TEMP", "TEMPDIR"}) {
      if (const char* path = getenv(env_var)) {
        tmp_dir = path;
        break;
      }
    }
    std::ostringstream oss;
    oss << tmp_dir << "/symm_mem-" << pid;
    return oss.str();
  }

  std::string socket_name_;
  int socket_;
};

constexpr size_t signal_pad_size = 2048;
const std::string store_comm_prefix = "CUDASymmetricMemory";

static size_t store_comm_seq_id = 0;

template <typename T>
std::vector<T> store_all_gather(
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size,
    T val) {
  static_assert(std::is_trivially_copyable_v<T>);

  std::vector<std::string> peer_keys;
  for (int r = 0; r < world_size; ++r) {
    std::ostringstream oss;
    oss << store_comm_prefix << "/" << store_comm_seq_id << "/" << r;
    peer_keys.push_back(oss.str());
  }
  ++store_comm_seq_id;

  {
    std::vector<uint8_t> payload(
        reinterpret_cast<uint8_t*>(&val),
        reinterpret_cast<uint8_t*>(&val) + sizeof(T));
    store->set(peer_keys[rank], payload);
  }

  std::vector<T> peer_vals;
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      peer_vals.push_back(val);
      continue;
    }
    store->wait({peer_keys[r]});
    auto payload = store->get(peer_keys[r]);
    TORCH_CHECK(payload.size() == sizeof(T));
    T peer_val{};
    std::memcpy(&peer_val, payload.data(), sizeof(T));
    peer_vals.push_back(peer_val);
  }
  return peer_vals;
}

void store_barrier(
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size) {
  store_all_gather(store, rank, world_size, 0);
}

void map_block(
    void** ptr,
    c10d::symmetric_memory::HandleType handle,
    size_t size,
    int device_idx) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto driver_api = c10::cuda::DriverAPI::get();
  auto dev_ptr = reinterpret_cast<hipDeviceptr_t*>(ptr);
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemAddressReserve_(dev_ptr, size, 0ULL, 0, 0ULL));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemMap_(*dev_ptr, size, 0, handle, 0ULL));

  hipMemAccessDesc desc;
  desc.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  desc.location.id = static_cast<int>(device_idx);
  desc.flags = hipMemAccessFlagsProtReadWrite;
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemSetAccess_(*dev_ptr, size, &desc, 1));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

} // namespace

namespace c10d {
namespace symmetric_memory {

AllocationRef::AllocationRef(void* ptr, HandleType handle, size_t block_size, int device_idx)
    : ptr(ptr), handle(handle), block_size(block_size), device_idx(device_idx) {}

AllocationRef::~AllocationRef() {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  // Leak the cuda allocations during static deinitialization
  if (is_finalizing()) {
    return;
  }
  auto driver_api = c10::cuda::DriverAPI::get();
  c10::cuda::CUDAGuard guard(device_idx);
  C10_CUDA_CHECK(hipDeviceSynchronize());
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemUnmap_(reinterpret_cast<hipDeviceptr_t>(ptr), block_size));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemRelease_(handle));
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

CUDASymmetricMemory::CUDASymmetricMemory(
    std::vector<c10::intrusive_ptr<AllocationRef>> alloc_refs,
    std::vector<void*> buffers,
    std::vector<void*> signal_pads,
    HandleType mc_handle,
    void* mc_addr,
    size_t buffer_size,
    int local_device_idx,
    int rank,
    int world_size)
    : alloc_refs_(std::move(alloc_refs)),
      buffers_(std::move(buffers)),
      signal_pads_(std::move(signal_pads)),
      mc_handle_(mc_handle),
      mc_addr_(mc_addr),
      buffer_size_(buffer_size),
      local_device_idx_(local_device_idx),
      rank_(rank),
      world_size_(world_size) {
  const size_t arr_size = sizeof(void*) * world_size_;
  buffers_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));
  signal_pads_dev_ = reinterpret_cast<void**>(
      c10::cuda::CUDACachingAllocator::raw_alloc(arr_size));

  c10::cuda::CUDAGuard guard(local_device_idx);
  AT_CUDA_CHECK(hipMemcpy(
      buffers_dev_, buffers_.data(), arr_size, hipMemcpyHostToDevice));
  AT_CUDA_CHECK(hipMemcpy(
      signal_pads_dev_, signal_pads_.data(), arr_size, hipMemcpyHostToDevice));
}

std::vector<void*> CUDASymmetricMemory::get_buffer_ptrs() {
  return buffers_;
}

std::vector<void*> CUDASymmetricMemory::get_signal_pad_ptrs() {
  return signal_pads_;
}

void** CUDASymmetricMemory::get_buffer_ptrs_dev() {
  return buffers_dev_;
}

void** CUDASymmetricMemory::get_signal_pad_ptrs_dev() {
  return signal_pads_dev_;
}

size_t CUDASymmetricMemory::get_buffer_size() {
  return buffer_size_;
}

size_t CUDASymmetricMemory::get_signal_pad_size() {
  return signal_pad_size;
}

bool CUDASymmetricMemory::has_multicast_support() {
  return mc_addr_ != nullptr;
}

void* CUDASymmetricMemory::get_multicast_ptr() {
  return mc_addr_;
}

at::Tensor CUDASymmetricMemory::get_buffer(
    int rank,
    c10::IntArrayRef sizes,
    c10::ScalarType dtype,
    int64_t storage_offset) {
  const size_t numel = std::accumulate(
      sizes.begin(),
      sizes.end(),
      static_cast<size_t>(1),
      std::multiplies<size_t>());
  const auto element_size = c10::elementSize(dtype);
  const auto req_size = (numel + storage_offset) * element_size;
  TORCH_CHECK(
      req_size <= buffer_size_,
      "CUDASymmetricMemory::get_buffer: the requested size (",
      req_size,
      " bytes) exceeds the allocated size (",
      buffer_size_,
      " bytes)");
  auto data_ptr = reinterpret_cast<uint8_t*>(buffers_[rank]) +
      storage_offset * element_size;
  auto device = c10::Device(c10::DeviceType::CUDA, local_device_idx_);
  auto options = at::TensorOptions().dtype(dtype).device(device);
  return at::for_blob(data_ptr, sizes)
      .options(options)
      .target_device(device)
      .make_tensor();
}

at::Tensor CUDASymmetricMemory::get_signal_pad(
    int rank,
    c10::IntArrayRef sizes,
    std::optional<c10::ScalarType> dtype,
    int64_t storage_offset) {
  // If the dtype is unspecified, default it to UInt32, as it
  // is the most common type for signaling purposes.
  if (!dtype.has_value()) {
    dtype = c10::ScalarType::UInt32;
  }

  // If the shape is unspecified, treat the signal pad as a 1d tensor.
  const auto element_size = c10::elementSize(*dtype);
  std::vector<int64_t> shape;
  if (sizes.size() != 0) {
    shape = sizes.vec();
  } else {
    shape.push_back(signal_pad_size / element_size);
  }

  const size_t numel = std::accumulate(
      shape.begin(),
      shape.end(),
      static_cast<size_t>(1),
      std::multiplies<size_t>());
  const auto req_size = (numel + storage_offset) * element_size;
  TORCH_CHECK(
      req_size <= signal_pad_size,
      "CUDASymmetricMemory::get_signal_pad: the requested size (",
      req_size,
      " bytes) exceeds the allocated size (",
      signal_pad_size,
      " bytes)");
  auto data_ptr = reinterpret_cast<uint8_t*>(signal_pads_[rank]) +
      storage_offset * element_size;
  auto device = c10::Device(c10::DeviceType::CUDA, local_device_idx_);
  auto options = at::TensorOptions().dtype(*dtype).device(device);
  return at::for_blob(data_ptr, shape)
      .options(options)
      .target_device(device)
      .make_tensor();
}

void check_channel(int channel, int world_size) {
  TORCH_CHECK(
      channel >= 0,
      "channel for barrier(), put_signal() and wait_signal() ",
      "must be greater than 0 (got ",
      channel,
      ")");
  const size_t num_channels = signal_pad_size / sizeof(uint32_t) * world_size;
  TORCH_CHECK(
      static_cast<size_t>(channel) < num_channels,
      "The maximum supported channel for barrier(), put_signal() and wait_signal() is ",
      num_channels - 1,
      " (got ",
      channel,
      ")");
}

static __global__ void barrier_kernel(
    uint32_t** signal_pads,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x < world_size) {
    auto target_rank = threadIdx.x;
    if (target_rank == rank) {
      return;
    }
    auto put_success = try_put_signal<std::memory_order_release>(
        signal_pads[target_rank] + world_size * channel + rank, timeout_ms);
    if (!put_success) {
      printf(
          "[FATAL] CUDASymmetricMemory::barrier: rank %d failed to send signal "
          "to rank %d on channel %d after %lu microseconds\n",
          rank,
          target_rank,
          channel,
          timeout_ms);
      trap();
    }
    auto wait_success = try_wait_signal<std::memory_order_acquire>(
        signal_pads[rank] + world_size * channel + target_rank, timeout_ms);
    if (!wait_success) {
      printf(
          "[FATAL] CUDASymmetricMemory::barrier: rank %d failed to receive signal "
          "from rank %d on channel %d after %lu microseconds\n",
          rank,
          target_rank,
          channel,
          timeout_ms);
      trap();
    }
  }
}

void CUDASymmetricMemory::barrier(int channel, size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  barrier_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void put_signal_kernel(
    uint32_t** signal_pads,
    int dst_rank,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x == 0) {
    bool success = try_put_signal<std::memory_order_release>(
        signal_pads[dst_rank] + world_size * channel + rank, timeout_ms);
    if (!success) {
      printf(
          "[FATAL] CUDASymmetricMemory::put_signal: rank %d failed to send signal "
          "to rank %d on channel %d after %lu microseconds\n",
          rank,
          dst_rank,
          channel,
          timeout_ms);
      trap();
    }
  }
}

void CUDASymmetricMemory::put_signal(
    int dst_rank,
    int channel,
    size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  put_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      dst_rank,
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void wait_signal_kernel(
    uint32_t** signal_pads,
    int src_rank,
    int channel,
    int rank,
    int world_size,
    size_t timeout_ms) {
  if (threadIdx.x == 0) {
    bool success = try_wait_signal<std::memory_order_acquire>(
        signal_pads[rank] + world_size * channel + src_rank, timeout_ms);
    if (!success) {
      printf(
          "[FATAL] CUDASymmetricMemory::wait_signal rank %d failed to receive signal "
          "from rank %d on channel %d after %lu microseconds\n",
          rank,
          src_rank,
          channel,
          timeout_ms);
#if !defined(USE_ROCM)
      __trap();
#else
      assert(0);
#endif
    }
  }
  __threadfence_system();
}

void CUDASymmetricMemory::wait_signal(
    int src_rank,
    int channel,
    size_t timeout_ms) {
  check_channel(channel, world_size_);
  c10::cuda::CUDAGuard guard(local_device_idx_);
  wait_signal_kernel<<<1, C10_WARP_SIZE, 0, at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<uint32_t**>(signal_pads_dev_),
      src_rank,
      channel,
      rank_,
      world_size_,
      timeout_ms);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

int CUDASymmetricMemory::get_rank() {
  return rank_;
}

int CUDASymmetricMemory::get_world_size() {
  return world_size_;
}

Block::Block(
  c10::intrusive_ptr<AllocationRef> alloc_ref,
  int device_idx,
  size_t block_size,
  size_t buffer_size,
  size_t signal_pad_offset,
  const std::optional<std::string>& group_name)
    : alloc_ref(std::move(alloc_ref)),
      device_idx(device_idx),
      block_size(block_size),
      buffer_size(buffer_size),
      signal_pad_offset(signal_pad_offset),
      default_group_name(std::move(group_name)) {}

void* CUDASymmetricMemoryAllocator::alloc(
    size_t size,
    int device_idx,
    const std::optional<std::string>& group_name) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  c10::cuda::CUDAGuard guard(device_idx);
  device_idx = static_cast<int>(guard.current_device().index());

  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  // NOLINTNEXTLINE(bugprone-signed-char-misuse)
  prop.location.id = device_idx;
  prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;

  size_t signal_pad_offset = at::round_up(size, 16UL);
  size_t block_size = signal_pad_offset + signal_pad_size;

  size_t granularity;
  auto driver_api = c10::cuda::DriverAPI::get();
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemGetAllocationGranularity_(
      &granularity, &prop, hipMemAllocationGranularityRecommended));
  block_size = at::round_up(block_size, granularity);

  HandleType handle;
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMemCreate_(&handle, block_size, &prop, 0));

  void* ptr = nullptr;
  map_block(&ptr, handle, block_size, device_idx);

  AT_CUDA_CHECK(hipMemset(ptr, 0, block_size));

  auto alloc_ref = c10::make_intrusive<AllocationRef>(ptr, handle, block_size, device_idx);
  auto block = c10::make_intrusive<Block>(
      std::move(alloc_ref),
      device_idx,
      block_size,
      size,
      signal_pad_offset,
      group_name);
  {
    std::unique_lock lock(mutex_);
    ptr_to_block_.emplace(ptr, std::move(block));
  }
  return ptr;
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

void CUDASymmetricMemoryAllocator::free(void* ptr) {
  std::unique_lock lock(mutex_);
  ptr_to_block_.erase(ptr);
}

size_t CUDASymmetricMemoryAllocator::get_alloc_size(void* ptr) {
  auto block = find_block(ptr);
  TORCH_CHECK(
      block != nullptr,
      "CUDASymmetricMemoryAllocator::get_alloc_size: input must be allocated ",
      "via CUDASymmetricMemoryAllocator::alloc");
  return block->buffer_size;
}

struct RendezvousRequest {
  int device_idx;
  int pid;
  size_t block_size;
  size_t buffer_size;
  size_t signal_pad_offset;
  bool has_multicast_support;
};

void validate_rendezvous_requests(
    const std::vector<RendezvousRequest>& reqs,
    int world_size) {
  TORCH_CHECK(reqs.size() == (size_t)world_size);

  std::unordered_set<int> device_indices;
  device_indices.reserve(world_size);
  for (auto req : reqs) {
    device_indices.insert(req.device_idx);
  }
  if (!allow_overlapping_devices() &&
      device_indices.size() < (size_t)world_size) {
    TORCH_CHECK(
        false,
        "CUDASymmetricMemoryAllocator::rendezvous: ",
        "detected allocations from overlapping devices ",
        "from different ranks.");
  }

  for (int r = 1; r < world_size; ++r) {
    TORCH_CHECK(reqs[r].block_size == reqs[0].block_size);
    TORCH_CHECK(reqs[r].buffer_size == reqs[0].buffer_size);
    TORCH_CHECK(reqs[r].signal_pad_offset == reqs[0].signal_pad_offset);
  }
}

static bool check_group_multicast_support(
    const std::vector<RendezvousRequest>& reqs) {
  std::vector<size_t> ranks_with_multicast_support;
  for (size_t r = 0; r < reqs.size(); ++r) {
    if (reqs[r].has_multicast_support) {
      ranks_with_multicast_support.push_back(r);
    }
  }
  if (ranks_with_multicast_support.size() == reqs.size()) {
    return true;
  } else {
    // We don't expect this to happen. But we want to let the user to know if
    // this happens.
    if (ranks_with_multicast_support.size() != 0) {
      LOG(WARNING)
          << "Only a subset of ranks in the group has multicast support: "
          << ranks_with_multicast_support << " (world_size=" << reqs.size()
          << "). Skipping multicast initialization because this is unexpected.";
    }
    return false;
  }
}

static void init_multicast_for_block(
    HandleType& mc_handle,
    void*& mc_addr,
    const c10::intrusive_ptr<Block>& block,
    IpcChannel& ipc_channel,
    const std::vector<int>& pids,
    const c10::intrusive_ptr<c10d::Store>& store,
    int rank,
    int world_size) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED) && \
    defined(CUDART_SUPPORTS_MULTICAST)
  auto driver_api = c10::cuda::DriverAPI::get();
  if (rank == 0) {
    CUmulticastObjectProp mc_prop{};
    mc_prop.numDevices = world_size;
    mc_prop.handleTypes = hipMemHandleTypePosixFileDescriptor;
    mc_prop.size = block->block_size;

    auto err = driver_api->cuMulticastCreate_(&mc_handle, &mc_prop);
    if (err != hipSuccess) {
      const char* err_str;
      hipError_t get_error_str_err = driver_api->cuGetErrorString_(err, &err_str);
      if (get_error_str_err != hipSuccess) {
        err_str = "unknown cuda driver error";
      }
      LOG(WARNING)
          << "SymmetricMemory: cuMulticastCreate failed with: \"" << err_str
          << "\". Gracefully skipping multicast initialization. "
          << "However, this is unexpected. Please report the issue on GitHub.";
      // Allow peers gracefully skip multicast initialization by sending -1
      ipc_channel.broadcast_fds(rank, 0, pids, -1);
      return;
    }

    int mc_fd;
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
        &mc_fd, mc_handle, hipMemHandleTypePosixFileDescriptor, 0));
    ipc_channel.broadcast_fds(rank, 0, pids, mc_fd);
    // Ref count is incremented as soon as SCM_RIGHTS send happens
    close(mc_fd);
  } else {
    int mc_fd = ipc_channel.broadcast_fds(rank, 0, pids, -1);
    if (mc_fd == -1) {
      return;
    }
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
        &mc_handle,
        (void*)(uintptr_t)mc_fd,
        hipMemHandleTypePosixFileDescriptor));
    close(mc_fd);
  }

  // All rank adds their physical allocation to the multicast object
  C10_CUDA_DRIVER_CHECK(
      driver_api->cuMulticastAddDevice_(mc_handle, block->device_idx));
  C10_CUDA_DRIVER_CHECK(driver_api->cuMulticastBindMem_(
      mc_handle, 0, block->alloc_ref->handle, 0, block->block_size, 0));

  map_block(&mc_addr, mc_handle, block->block_size, block->device_idx);
  store_barrier(store, rank, world_size);
#endif
}

c10::intrusive_ptr<SymmetricMemory> CUDASymmetricMemoryAllocator::rendezvous(
    void* ptr,
    const std::optional<std::string>& group_name) {
#if !defined(USE_ROCM) && defined(PYTORCH_C10_DRIVER_API_SUPPORTED)
  auto block = find_block(ptr);
  if (block == nullptr) {
    return nullptr;
  }

  // The group_name passed to rendezvous() takes precedence over
  // the default group_name specified during allocation.
  std::string group_name_;
  if (group_name.has_value()) {
    group_name_ = *group_name;
  } else {
    if (!block->default_group_name.has_value()) {
      TORCH_CHECK(
          false,
          "CUDASymmetricMemory::rendezvous: `group_name` is neither "
          "specified during allocation nor passed to rendezvous().");
    }
    group_name_ = *block->default_group_name;
  }

  auto it = block->symm_mems.find(group_name_);
  if (it != block->symm_mems.end()) {
    return it->second;
  }

  c10::cuda::CUDAGuard guard(block->device_idx);

  IpcChannel ipc_channel;
  auto group_info = get_group_info(group_name_);
  auto store = group_info.store;
  int rank = group_info.rank;
  int world_size = group_info.world_size;

  auto driver_api = c10::cuda::DriverAPI::get();
  int block_fd;
  C10_CUDA_DRIVER_CHECK(driver_api->cuMemExportToShareableHandle_(
      &block_fd,
      block->alloc_ref->handle,
      hipMemHandleTypePosixFileDescriptor,
      0));

  auto local_req = RendezvousRequest{
      .device_idx = block->device_idx,
      .pid = getpid(),
      .block_size = block->block_size,
      .buffer_size = block->buffer_size,
      .signal_pad_offset = block->signal_pad_offset,
      .has_multicast_support = device_has_multicast_support(block->device_idx)};
  auto reqs = store_all_gather(store, rank, world_size, local_req);
  validate_rendezvous_requests(reqs, world_size);

  std::vector<int> pids(world_size);
  for (int r = 0; r < world_size; ++r) {
    pids[r] = reqs[r].pid;
  }
  auto imported_fds = ipc_channel.all_gather_fds(rank, pids, block_fd);

  std::vector<HandleType> handles(world_size);
  std::vector<void*> buffers(world_size, nullptr);
  std::vector<void*> signal_pads(world_size, nullptr);

  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      handles[r] = block->alloc_ref->handle;
      buffers[r] = ptr;
      signal_pads[r] = (void*)((uintptr_t)ptr + block->signal_pad_offset);
      continue;
    }
    C10_CUDA_DRIVER_CHECK(driver_api->cuMemImportFromShareableHandle_(
        &handles[r],
        (void*)(uintptr_t)imported_fds[r],
        hipMemHandleTypePosixFileDescriptor));
    map_block(&buffers[r], handles[r], block->block_size, block->device_idx);
    signal_pads[r] = (void*)((uintptr_t)buffers[r] + block->signal_pad_offset);
    close(imported_fds[r]);
  }
  store_barrier(store, rank, world_size);
  close(block_fd);

  HandleType mc_handle{};
  void* mc_addr = nullptr;
  bool group_has_multicast_support = check_group_multicast_support(reqs);
  if (!allow_overlapping_devices() && group_has_multicast_support) {
    init_multicast_for_block(
        mc_handle, mc_addr, block, ipc_channel, pids, store, rank, world_size);
  }

  std::vector<c10::intrusive_ptr<AllocationRef>> alloc_refs;
  for (int r = 0; r < world_size; ++r) {
    if (r == rank) {
      alloc_refs.emplace_back(block->alloc_ref);
      continue;
    }
    alloc_refs.push_back(c10::make_intrusive<AllocationRef>(
        buffers[r], handles[r], block->block_size, block->device_idx));
  }

  auto symm_mem = c10::make_intrusive<CUDASymmetricMemory>(
      std::move(alloc_refs),
      std::move(buffers),
      std::move(signal_pads),
      mc_handle,
      mc_addr,
      block->buffer_size,
      block->device_idx,
      group_info.rank,
      group_info.world_size);
  block->symm_mems[group_name_] = symm_mem;
  return symm_mem;
#else
  TORCH_CHECK(
      false, "CUDASymmetricMemory requires PYTORCH_C10_DRIVER_API_SUPPORTED");
#endif
}

bool CUDASymmetricMemoryAllocator::has_multicast_support(int device_idx) {
  return device_has_multicast_support(device_idx);
}

c10::intrusive_ptr<Block> CUDASymmetricMemoryAllocator::find_block(void* ptr) {
  std::shared_lock lock(mutex_);
  auto it = ptr_to_block_.find(ptr);
  if (it == ptr_to_block_.end()) {
    return nullptr;
  }
  return it->second;
}

struct RegisterCUDASymmetricMemoryAllocator {
  RegisterCUDASymmetricMemoryAllocator() {
    register_allocator(
        c10::DeviceType::CUDA,
        c10::make_intrusive<CUDASymmetricMemoryAllocator>());
  }
};

static RegisterCUDASymmetricMemoryAllocator register_allocator_;

} // namespace symmetric_memory
} // namespace c10d
