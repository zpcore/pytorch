
#include <hip/hip_runtime.h>
#ifdef USE_C10D_NCCL

#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/torch.h>
#include <algorithm>
#include <torch/csrc/distributed/c10d/NanCheck.hpp>

namespace c10d {

// CUDA kernel to check if data has NAN, device side assert
// is raised if NAN is found
template <typename T>
__global__ void checkForNaN(T* data, size_t size) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < size; i += stride) {
    CUDA_KERNEL_ASSERT(!isnan(data[i]));
  }
}

// CHECK if a Tensor contains NAN in any of its element
void checkForNan(const at::Tensor& tensor, at::cuda::CUDAStream& stream) {
  // skip check for non float types
  if (!torch::is_floating_point(tensor)) {
    return;
  }
  const size_t maxNumThreadsPerBlock = 256;
  const size_t maxNumBlocks = 24;
  const size_t numThreadsPerBlock =
      std::min<size_t>(maxNumThreadsPerBlock, tensor.numel());

  const size_t numBlocks = std::min<size_t>(
      maxNumBlocks,
      (tensor.numel() + numThreadsPerBlock - 1) / numThreadsPerBlock);

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      tensor.scalar_type(),
      "checkForNaN",
      [&] {
        checkForNaN<scalar_t><<<numBlocks, numThreadsPerBlock, 0, stream>>>(
            tensor.data_ptr<scalar_t>(), tensor.numel());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace c10d

#endif // USE_C10D_NCCL
