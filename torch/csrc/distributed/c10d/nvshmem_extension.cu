#include "hip/hip_runtime.h"
#include <torch/csrc/distributed/c10d/nvshmem_extension.cuh>

#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemoryUtils.hpp>
#include <torch/csrc/distributed/c10d/SymmetricMemory.hpp>

#include <cuda_awbarrier_primitives.h>
// Use torch's cub wrapper instead of CUDA's <hipcub/hipcub.hpp>, see #55292
#include <ATen/cuda/cub.cuh>
#include <nvshmem.h>

namespace c10d::nvshmem_extension {

using c10d::symmetric_memory::StoreExchange;
static StoreExchange storeExchange = StoreExchange("nvshmem_ext");

#define THREADS_PER_BLOCK 512

constexpr int MiB = 1024 * 1024;

// Bootstrap based on user's setting for NCCL
// Long term, this may be a bit unclean; short term, it improves UX
void maybe_initialize_env_vars() {
  auto nccl_socket_if_name = c10::utils::get_env("NCCL_SOCKET_IFNAME");
  auto nccl_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nccl_ib_gid_index = c10::utils::get_env("NCCL_IB_GID_INDEX");
  auto nvshmem_socket_if_name =
      c10::utils::get_env("NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME");
  auto nvshmem_hca_list = c10::utils::get_env("NCCL_IB_HCA");
  auto nvshmem_ib_gid_index = c10::utils::get_env("NVSHMEM_IB_GID_INDEX");

  if (!nvshmem_socket_if_name.has_value() && nccl_socket_if_name.has_value()) {
    c10::utils::set_env(
        "NVSHMEM_BOOTSTRAP_UID_SOCK_IFNAME", nccl_socket_if_name->c_str());
  }
  if (!nvshmem_hca_list.has_value() && nccl_hca_list.has_value()) {
    c10::utils::set_env("NVSHMEM_ENABLE_NIC_PE_MAPPING", "1");
    c10::utils::set_env("NVSHMEM_HCA_LIST", nccl_hca_list->c_str());
  }
  if (!nvshmem_ib_gid_index.has_value() && nccl_ib_gid_index.has_value()) {
    c10::utils::set_env("NVSHMEM_IB_GID_INDEX", nccl_ib_gid_index->c_str());
  }
}

void initialize_nvshmem_with_store(
    c10::intrusive_ptr<c10d::Store> store,
    int rank,
    int world_size) {
  static bool is_initialized = false;
  if (is_initialized) {
    return;
  }

  maybe_initialize_env_vars();

  nvshmemx_uniqueid_t unique_id;
  TORCH_CHECK(
      nvshmemx_get_uniqueid(&unique_id) == 0, "nvshmemx_get_uniqueid failed");

  // Using an existing store_all_gather due to laziness.
  // TODO(yifu): should use broadcast
  auto unique_ids = storeExchange.all_gather(store, rank, world_size, unique_id);

  nvshmemx_init_attr_t attr;
  nvshmemx_set_attr_uniqueid_args(rank, world_size, &unique_ids[0], &attr);

  TORCH_CHECK(
      nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr) == 0,
      "nvshmemx_init_attr failed");

  is_initialized = true;
}

void* nvshmem_malloc(size_t size) {
  return ::nvshmem_malloc(size);
}

void* nvshmem_ptr(const void* dest, int pe) {
  return ::nvshmem_ptr(dest, pe);
}

std::unordered_map<std::string, nvshmem_team_t> group_name_to_team_;

nvshmem_team_t group_to_team(
    const std::string& group_name,
    const std::vector<int>& global_ranks) {
  auto it = group_name_to_team_.find(group_name);
  if (it != group_name_to_team_.end()) {
    return it->second;
  }
  TORCH_CHECK(global_ranks.size() > 1);
  int stride = global_ranks[1] - global_ranks[0];
  for (size_t r = 1; r < global_ranks.size(); ++r) {
    TORCH_CHECK(global_ranks[r] - global_ranks[r - 1] == stride);
  }

  nvshmem_team_t team;
  TORCH_CHECK(
      nvshmem_team_split_strided(
          NVSHMEM_TEAM_WORLD,
          global_ranks[0],
          stride,
          global_ranks.size(),
          nullptr,
          0,
          &team) == 0);
  group_name_to_team_[group_name] = team;
  TORCH_CHECK(team != NVSHMEM_TEAM_INVALID);
  return team;
}

at::Tensor nvshmem_broadcast(at::Tensor& input, const std::string& group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());
  void* buffer_ptr = input_hdl->get_buffer_ptrs()[rank];

  auto stream = at::cuda::getCurrentCUDAStream();
  nvshmemx_broadcastmem_on_stream(team, buffer_ptr, buffer_ptr, input_hdl->get_buffer_size(), 0, stream);
  return input;
}

at::Tensor nvshmem_all_to_all(
    at::Tensor& input,
    at::Tensor& out,
    std::string group_name) {
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();
  auto team = group_to_team(group_name, input_hdl->get_rank_to_global_rank());

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  size_t bytes_per_rank = input_hdl->get_buffer_size() / world_size;

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());
  nvshmemx_alltoallmem_on_stream(team, output_ptr, input_ptr, bytes_per_rank, stream);
  return out;
}

// This is an exclusive prefix sum function that calculates read (or write) offsets for each peer.
__device__ void prefixSum(int64_t *odata, int64_t *idata, int n) {
  // Specialize BlockScan for a 1D block of threads, of type int64_t.
  // - `BLOCK_SCAN_WARP_SCANS` is a low-latency scan algorithm (instead of high
  // throughput which we don't need here).
  // - `at_cuda_detail::cub` is torch's cub wrapper, see #55292.
  using BlockScanT = at_cuda_detail::hipcub::BlockScan<int64_t, THREADS_PER_BLOCK, at_cuda_detail::hipcub::BLOCK_SCAN_WARP_SCANS>;
  // Allocate shared memory for BlockScan
  __shared__ typename BlockScanT::TempStorage temp_storage;

  // TODO: currently it is assumed that the number of PE's is smaller than
  // `THREADS_PER_BLOCK`
  CUDA_KERNEL_ASSERT(n <= THREADS_PER_BLOCK);

  // Obtain input item for each thread
  int tid = threadIdx.x;
  int64_t thread_data = (tid < n) ? idata[tid] : 0;

  // Collectively compute the block-wide exclusive prefix sum
  BlockScanT(temp_storage).ExclusiveSum(thread_data, thread_data);

  // Store the result
  if (tid < n) {
    odata[tid] = thread_data;
  }
}

// This kernel is used to exchange output splits and source offsets between peers.
// `in_out_splits` is of size (3, npes) and contains:
// - input splits (IN)
// - output splits (OUT) and
// - source offsets (OUT).
__global__ void exchangeSplitAndOffset(int64_t* in_out_splits, int mype, int npes) {
  auto input_splits = in_out_splits;
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int tid = threadIdx.x;

  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];

  // Scan input splits to get the source offsets
  prefixSum(peer_offsets, input_splits, npes);
  __syncthreads();;

  // Use 1 block to do the exchange
  if (tid < npes) {
    int peer = tid;
    nvshmem_int64_p(source_offsets + mype, peer_offsets[peer], peer);
    nvshmem_int64_p(output_splits + mype, input_splits[peer], peer);
  }
  // This barrier ensures that all remote PEs see the updated values
  nvshmemx_barrier_all_block();
}

// This kernel is used to do the actual data exchange.
// `in_out_splits` has the same definition as in `exchangeSplitAndOffset`.
// `stride` is the stride at dim 0, unit in byte.
__global__ void allToAllV(void *send_data, void *recv_data, int64_t* in_out_splits, size_t stride, int mype, int npes) {
  auto output_splits = in_out_splits + npes;
  auto source_offsets = in_out_splits + npes * 2;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blocks_per_peer = max(gridDim.x / npes, 1);

  // Calculate the output offsets
  __shared__ int64_t peer_offsets[THREADS_PER_BLOCK];
  prefixSum(peer_offsets, output_splits, npes);
  __syncthreads();

  // Target a different peer based on bid
  for (int i = bid / blocks_per_peer; i < npes; i += gridDim.x / blocks_per_peer) {
    int peer = (mype + i) % npes;
    // Total amount from `peer`
    auto peer_size = output_splits[peer] * stride;
    // Amount to get from `peer` in this block
    auto block_size = peer_size / blocks_per_peer;
    // Being lazy here, we should handle the residual if the division is not exact
    CUDA_KERNEL_ASSERT(block_size * blocks_per_peer == peer_size);
    // This block's offset in the data from `peer`
    auto block_offset = block_size * (bid % blocks_per_peer);
    auto source_offset = source_offsets[peer] * stride + block_offset;
    auto write_offset = peer_offsets[peer] * stride + block_offset;
    nvshmemx_getmem_block(
      (char*)recv_data + write_offset,
      (char*)send_data + source_offset,
      block_size,
      peer);
  }
  // Write out the output offsets (to the scratchpad line)
  if (bid == 0 && tid < npes) {
    source_offsets[tid] = peer_offsets[tid];
  }
}

at::Tensor nvshmem_all_to_all_vdev(
    at::Tensor& input,
    at::Tensor& out,
    at::Tensor& in_out_splits,
    std::string group_name) {
  /* Perform AllToAllv operation using NVSHMEM, with split information provided on device.
   * Arguments:
   *  - `input` is the input tensor
   *  - `out` is the output tensor
   *  - `in_out_splits` is a 2D tensor of size (3, npes). The rows are (in order):
        input splits (IN)
        output splits (OUT) and
        output offsets (OUT).
  */
  auto input_hdl = c10d::symmetric_memory::rendezvous(input, group_name);
  auto out_hdl = c10d::symmetric_memory::rendezvous(out, group_name);
  auto splits_hdl = c10d::symmetric_memory::rendezvous(in_out_splits, group_name);
  int rank = input_hdl->get_rank();
  int world_size = input_hdl->get_world_size();

  void* input_ptr = input_hdl->get_buffer_ptrs()[rank];
  void* output_ptr = out_hdl->get_buffer_ptrs()[rank];
  int64_t* splits_ptr = (int64_t*)(splits_hdl->get_buffer_ptrs()[rank]);

  auto stream = at::cuda::getCurrentCUDAStream(input.device().index());

  // Exchange output splits and source offsets
  // Use collective launch because kernel involves nvshmem barrier
  void* args0[] = {
      &splits_ptr,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)exchangeSplitAndOffset,
      dim3(1),
      dim3(THREADS_PER_BLOCK),
      args0,
      0,
      stream);

  // CTA Tuning
  // Intra-node: use multiple blocks per peer to increase data parallelism, up to 8.
  // Up to 1 MB -> 1 block
  // Up to 2 MB -> 2 blocks
  // Up to 4 MB -> 4 blocks
  // More -> 8 blocks
  // The tuning for `num_blocks` below multiplies these numbers by world_size
  // (e.g. 8 -> 8 * 8). If world_size is smaller, we simply shift the blocks
  // towards data parallelism. (There may be room for improvement here)
  auto input_size = input.numel() * input.element_size();
  int num_blocks = input_size < MiB ? 8 :
      (input_size < 2 * MiB ? 16 :
      (input_size < 4 * MiB ? 32 : 64));

  // Inter-node: limit the total the number of blocks to 8 which is able to
  // drive 57 GB/s bandwidth in test, enough to drive a 400 Gb/s NIC.
  // TODO: better intra vs inter detection, currently it is based on world_size
  if (world_size > 8) {
    num_blocks = std::min(num_blocks, 8);
  }

  // Stride at dim 0 (assuming input is contiguous, TODO)
  size_t stride_bytes = input.stride(0) * input.element_size();

  // All to all data exchange
  void* args1[] = {
      &input_ptr,
      &output_ptr,
      &splits_ptr,
      &stride_bytes,
      &rank,
      &world_size};
  nvshmemx_collective_launch(
      (const void*)allToAllV,
      dim3(num_blocks),
      dim3(THREADS_PER_BLOCK),
      args1,
      0,
      stream);
  return out;
}

} // namespace c10d::nvshmem_extension


TORCH_LIBRARY_IMPL(symm_mem, CUDA, m) {
  m.impl("nvshmem_broadcast", c10d::nvshmem_extension::nvshmem_broadcast);
  m.impl("nvshmem_all_to_all", c10d::nvshmem_extension::nvshmem_all_to_all);
  m.impl("nvshmem_all_to_all_vdev", c10d::nvshmem_extension::nvshmem_all_to_all_vdev);
}
